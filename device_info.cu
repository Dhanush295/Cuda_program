#include<iostream>

using namespace std;

int main(){

    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount( &count);

    for(int i=0; i<count; i++){
        hipGetDeviceProperties(&prop, i);
        cout<<" ----------GEneral Information for device "<< i << " ---\n";
        cout<<"Name: " << prop.name<<'\n';

        cout<<"Compute capability: " << prop.major, prop.minor<<'\n';

        cout<<"Clock rate: " << prop.clockRate<<'\n';

        cout<<"Device copy overlap: \n" ;

        if(prop.deviceOverlap){
            cout<<"Enabled \n";
        }
        else{
                cout<<"Disabled \n";
            }

        cout<<"Kernal execution timeout: \n" ;

        if(prop.kernelExecTimeoutEnabled){
            cout<<"Enabled \n";
        }
        else{
                cout<<"Disabled \n";
            }


        cout<<" ----------Memory information for device "<< i << " -----\n";
        cout<<"Total global mem: " << prop.totalGlobalMem<<'\n';

        cout<<"Total constant mem: " << prop.totalConstMem<<'\n';

        cout<<"Max mem pitch: " << prop.memPitch<<'\n';

        cout<<"Texture Alignment: " << prop.textureAlignment<<'\n';

        ///////////////////////////////////////////

        cout<<" ----------MP information for device "<< i << " -----\n";
        cout<<"Multiprocessor count: " << prop.multiProcessorCount<<'\n';

        cout<<"shared mem per mp: " << prop.sharedMemPerBlock<<'\n';

        cout<<"Register per mp: " << prop.regsPerBlock<<'\n';

        cout<<"Threads in wrap: " << prop.warpSize<<'\n';



        cout<<"Max thread per block: " << prop.maxThreadsPerBlock<<'\n';

        cout<<"Max thread dimensions: " << prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]<<'\n';

        cout<<"Max grid dimensions: " << prop.maxGridSize[0],  prop.maxGridSize[1], prop.maxGridSize[2]<<'\n';

    }


    return 0;
}