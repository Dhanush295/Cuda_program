
#include <hip/hip_runtime.h>
#include<iostream>

#define TILE_SIZE 16

using namespace std;

__global__ void MatrixMultiShared(float* A, float* B, float* C, int N){
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    int row = threadIdx.y + blockIdx.y * TILE_SIZE;
    int col = threadIdx.x + blockIdx.x * TILE_SIZE;

    float val = 0.0f;

    for(int i =0; i < (N + TILE_SIZE -1)/ TILE_SIZE; i++){
        if(row < N && (i * TILE_SIZE + threadIdx.x) < N){
            tile_A[threadIdx.y][threadIdx.x] = A[row * N + i * TILE_SIZE + threadIdx.x];
        }
        else{
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;
        }


        if(col < N && (i * TILE_SIZE + threadIdx.y) < N){
            tile_B[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * N + col];
        }
        else{
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for(int j=0; j<TILE_SIZE; j++){
            val+= tile_A[threadIdx.y][j] * tile_B[j][threadIdx.x];
        }

        __syncthreads();
    }

    if(row < N && col < N){
        C[row * N + col] = val;
    }
}


int main(){
    int N = 1024;
    int size = N * N * sizeof(float);

    float* h_a = (float*)malloc(size);
    float* h_b = (float*)malloc(size);
    float* h_c = (float*) malloc(size);

    for(int i=0; i< N*N; i++){
        h_a[i] = 1.0f;
        h_b[i] = 1.0f;
    }

    float *d_a, *d_b, *d_c;

    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((N + TILE_SIZE - 1)/ TILE_SIZE, (N + TILE_SIZE - 1)/TILE_SIZE );

    MatrixMultiShared<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // for(int i=0; i<N; i++){
    //     for(int j=0; j< N; j++){
    //         cout<<h_c[i * N + j]<< " ";
    //     }
    //     cout<<endl;
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;

}