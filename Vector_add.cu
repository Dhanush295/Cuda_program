#include<iostream>
#include<hip/hip_runtime.h>

#define N  10
using namespace std;


__global__ void add( int* a, int* b, int* c){

    int tid = blockIdx.x;

    if(tid < N){
        c[tid] = a[tid] + b[tid];
    }

}

int main(){
    int a[N] , b[N], c[N];
    
    int *dev_a , *dev_b , *dev_c;

    for(int i=0; i<N; i++ ){
        a[i] = i+1;
        b[i] = i*i;
    }

    hipMalloc((void **) &dev_a , N*sizeof(int));
    hipMalloc((void **) &dev_b , N*sizeof(int));
    hipMalloc((void **) &dev_c , N*sizeof(int));

    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<N ;i++){
        cout<<"A + B "<<c[i]<<"\n";
        
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;

}