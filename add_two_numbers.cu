#include<iostream>
#include<hip/hip_runtime.h>



__global__ void add(int a, int b, int *c){
    *c = a + b;
}

int main(){

int c;
int *dev_c;

hipMalloc((void **) &dev_c, sizeof(int));

std::cout<<"After: "<<dev_c<<"\n";

add<<<1,1>>>(2,7, dev_c);



hipMemcpy(&c, dev_c, sizeof(int) , hipMemcpyDeviceToHost);


std::cout<<"2 + 7 is "<< c;

hipFree(dev_c);

};