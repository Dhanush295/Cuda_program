#include<iostream>
#include<vector>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

__global__ void add_vec(int* a, int* b, int* c, int n){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < n){
        c[tid] = a[tid] + b[tid];
    }

}

void vec(int n){
    vector<int> one(n);
    vector<int> two(n);
    vector<int> sum_of_two_cpu(n);
    vector<int> sum_of_two(n);

    int *dev_a, *dev_b, *dev_c;

    for(int i=0; i<n; i++){
        one[i] = i+1;
        two[i] = i+2;
    }

    auto start_cpu = high_resolution_clock::now();

    for (int i = 0; i < n; i++) {
        sum_of_two_cpu[i] = one[i] + two[i];
    }

    auto end_cpu = high_resolution_clock::now();

    auto duration_cpu = duration_cast<microseconds>(end_cpu - start_cpu);


    hipMalloc((void**) &dev_a, n*sizeof(int));
    hipMalloc((void**) &dev_b, n*sizeof(int));
    hipMalloc((void**) &dev_c, n*sizeof(int));

    hipMemcpy(dev_a, one.data(), n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, two.data(), n*sizeof(int) , hipMemcpyHostToDevice);

    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);
    

    int threads_per_block = 256;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    add_vec<<<blocks_per_grid, threads_per_block>>>(dev_a, dev_b, dev_c, n);

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    float duration_gpu = 0;
    hipEventElapsedTime(&duration_gpu, start_gpu, stop_gpu);


    hipMemcpy(sum_of_two.data(), dev_c, n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    cout << "Time taken by CPU: " << duration_cpu.count() << " microseconds" << endl;
    cout << "Time taken by GPU: " << duration_gpu << " milliseconds" << endl;

    
    hipEventDestroy(start_gpu);
    hipEventDestroy(stop_gpu);

}


int main(){

    vector<int> a ={100, 1000, 100000, 1000000};

    for(int i=0; i<a.size(); i++){
        cout << "Vector size: " << a[i] << endl;
        vec(a[i]);
        cout << endl;
    }
    return 0;
}