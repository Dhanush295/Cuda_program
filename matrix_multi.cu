
#include <hip/hip_runtime.h>
#include<iostream>

#define N 1024

using namespace std;

__global__ void MatrixMultiDevice(float* d_a, float* d_b, float* d_c, int width){

    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;

    if(row < width && col < width){
        float val = 0.0f;

        for(int k=0; k< width; k++){
            val += d_a[row * width + k] * d_b[k * width + col];
        }

        d_c[row* width + col] = val;
    }
}


void matrixMultiHost(float* h_a, float* h_b, float* h_c, int width){

    int size = width * width * sizeof(float);

    float* d_a;
    float* d_b;
    float* d_c;

    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b , h_b, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1)/ dimBlock.x, (width + dimBlock.y - 1)/ dimBlock.y);

    MatrixMultiDevice<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


}

int main(){
    int width = N;
    int size = width * width * sizeof(float);

    float* h_a = (float*)malloc(size);
    float* h_b = (float*)malloc(size);
    float* h_c = (float*)malloc(size);

    for(int i=0; i< width* width; i++){
        h_a[i] = 1.0f;
        h_b[i] = 1.0f;
    }

    matrixMultiHost(h_a, h_b, h_c, width);

    // for(int i=0; i< width; i++){
    //     for(int j=0; j< width; j++){
    //         cout<<h_c[i * width + j] <<" ";
    //     }
    //     cout<<endl;
    // }

    free(h_a);
    free(h_b);
    free(h_c);
}