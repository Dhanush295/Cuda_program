
#include <hip/hip_runtime.h>
#include<iostream>
#define n 10

using namespace std;
const int threadperblock = 256;


__global__ void dot_product(int *a, int *b, int *c){
    __shared__ float cache[threadperblock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp =0;

    while(tid < n){
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x/2;
    while(i!= 0){
        if(cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];

        __syncthreads();
        i /= 2;
    }

    if(cacheIndex == 0){
        c[blockIdx.x] = cache[0];
        }
}

int main(){

    int a[n], b[n], c[n], cpu_stored , stored_dot_val;
    int *dev_a, *dev_b, *dev_c;

    for(int i=0; i<n; i++){
        a[i] = i+2;
        b[i] = i+1;
    }

    cpu_stored = 0;

    for(int i=0; i<n; i++){
        cpu_stored += a[i] * b[i];
    }

    cout<<"CPU STored Value: "<<cpu_stored<<endl;



    hipMalloc( (void **) &dev_a, n*sizeof(int) );
    hipMalloc( (void **) &dev_b, n*sizeof(int) );
    hipMalloc((void **) &dev_c, n*sizeof(int));

    hipMemcpy(dev_a, a, n*sizeof(n), hipMemcpyHostToDevice );
    hipMemcpy(dev_b, b, n*sizeof(n), hipMemcpyHostToDevice );


    int blockpergrid = (n + threadperblock -1)/ threadperblock;

    dot_product<<<blockpergrid, threadperblock>>>(dev_a, dev_b, dev_c);

    hipMemcpy(&c, dev_c, n*sizeof(int), hipMemcpyDeviceToHost);

    stored_dot_val = 0;

    for( int i=0; i<blockpergrid; i++){
        stored_dot_val += c[i];
    }

    cout<<"Stored Value is: "<< stored_dot_val<<endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}